#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>

#include "/content/lodepng.h"

const int ERROR_EXIT_VALUE = -1;


__global__
void cudaBlur(unsigned char* originalVals, unsigned char* blurredVals, 
int width, int height) {
  int uid = 300 * blockIdx.x + threadIdx.x;
  
  int pixelX = uid % width;
  int pixelY = uid / width;

  if(pixelY == 0 || pixelY == height-1){
    return;
  }

  int i, sum, up, down;
  int p = 4*uid;
  for(i=0; i<4; i++){
    sum = 0;
    if((i+1)%4 == 0){
      blurredVals[p+i] = originalVals[p+i];
    }else{
      up = ((pixelY-1) * (width) + pixelX) * 4;
      sum+=(originalVals[up+i-4]+originalVals[up+i]+originalVals[up+i+4]);
      sum+= (originalVals[p+i-4]+originalVals[p+i]+originalVals[p+i+4]);
      down = ((pixelY+1) * (width) + pixelX) * 4;
      sum+=(originalVals[down+i-4]+originalVals[down+i]+originalVals[down+i+4]);
      blurredVals[p+i] = sum / 9;
    }
    
  }
}

int main (int argc, char* argv[]) {

	  char* fileName = "/content/5n5PNs3.png";
    if (argc > 1)
        fileName = argv[1];
    
    char* outputFileName = "/content/output.png";
    if (argc > 2)
    	outputFileName = argv[2];

    unsigned int width, height;
    unsigned int lodepng_error;

    unsigned char* cpuImg1DValues = (unsigned char*) malloc( sizeof(unsigned char) * width * height * 4 );
    lodepng_error = lodepng_decode32_file(&cpuImg1DValues, &width, &height, fileName);

    if (lodepng_error) {
      printf("Error decoding png file: '%u' '%s'\n", lodepng_error, lodepng_error_text(lodepng_error));
      exit(ERROR_EXIT_VALUE);
    }

    int imgSize = width * height * 4;  // totalImgPixels

printf ("%d %d %d",width, height, imgSize);

	  if (width <= 0 || height <= 0) {
        printf("Unable to decode image. Validate file and try again\n");
        exit(ERROR_EXIT_VALUE);
    }

    unsigned char* gpuInputImgVals;
    hipMalloc((void**) &gpuInputImgVals, sizeof(unsigned char) * imgSize);
    hipMemcpy(gpuInputImgVals, cpuImg1DValues, sizeof(unsigned char) * imgSize, hipMemcpyHostToDevice);

    unsigned char* gpuOutputImgVals;
    hipMalloc((void**) &gpuOutputImgVals, sizeof(unsigned char) * imgSize);

    cudaBlur<<<300, 300>>>(gpuInputImgVals, gpuOutputImgVals, width, height);

    unsigned char* cpuOutImg = (unsigned char*) malloc( sizeof(unsigned char) * imgSize );
    hipMemcpy(cpuOutImg, gpuOutputImgVals, sizeof(unsigned char) * imgSize, hipMemcpyDeviceToHost);

   	hipDeviceSynchronize();

    lodepng_error = lodepng_encode32_file(outputFileName, cpuOutImg, width, height);
      
    if (lodepng_error) {
      printf("Error encoding png file: '%u' '%s'\n", lodepng_error, lodepng_error_text(lodepng_error));
      exit(ERROR_EXIT_VALUE);
    }

    free(cpuImg1DValues);
    free(cpuOutImg);
    hipFree(gpuInputImgVals);
    hipFree(gpuOutputImgVals);
}